#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/OpMathType.h>
#include <ATen/ceil_div.h>
#include <ATen/native/ForeachUtils.h>
#include <ATen/cuda/DeviceUtils.cuh>
#include <ATen/native/cuda/ForeachFunctors.cuh>
#include <ATen/native/cuda/MultiTensorApply.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_foreach_norm_native.h>

#include <ATen/ops/empty.h>
#include <ATen/ops/zeros.h>
#endif

namespace at::native {

// _foreach_norm supports only L1, L2, and inf norm
enum class NormType { L1, L2, LInf };

// NOTE: This is a simple variant of TensorListMetadata in MultiTensorApply.cuh
// as we only need to track addresses for the lpnorm_cleanup function below.
// Why is this struct necessary? For the same reason the TensorListMetadata
// struct is necessary--which is to ferry static metadata to the CUDA kernel
// while complying with the 4kb size constraint. Since we only need to track
// addresses, we introduce this struct to be able to fit more Tensor pointers at
// a time, currently 400 empirically, compared to the much smaller values in
// depth_to_max_tensors. This way, we can launch fewer kernels for better
// performance.
//
// IF YOU USE THIS STRUCT, PLEASE ADD A ONE-OFF TEST IN test_foreach.py AS THIS
// IS CURRENTLY ONLY TESTED FOR _foreach_norm.
const size_t MAX_TENSORS_PER_KERNEL = 400;
struct TensorListAddresses {
  const void* addresses[MAX_TENSORS_PER_KERNEL];
};

template <
    typename T,
    NormType norm_type,
    int depth = 1,
    int r_args_depth = 1,
    int res_arg_index = 0>
struct LpNormFunctor {
  using opmath_t = typename at::opmath_type<T>;
  __device__ __forceinline__ void operator()(
      int chunk_size,
      TensorListMetadata<depth>& tl,
      opmath_t* output_per_tensor,
      const int max_chunks_per_tensor) {
    const auto tensor_loc = tl.block_to_tensor[blockIdx.x];
    const auto chunk_idx = tl.block_to_chunk[blockIdx.x];
    auto n = tl.numel_for_tensor[tensor_loc];

    T* x = (T*)tl.addresses[0][tensor_loc];
    x += chunk_idx * chunk_size;
    n -= chunk_idx * chunk_size;

    __shared__ opmath_t s_vals[512];
    opmath_t vals[kILP];
    T r_x[kILP];
    for (int64_t i = 0; i < kILP; i++) {
      vals[i] = opmath_t(0);
      r_x[i] = T(0);
    }

    if (n % kILP == 0 && (chunk_size & kILP) == 0 && is_aligned(x)) {
      for (int64_t i_start = threadIdx.x;
           i_start * kILP < n && i_start * kILP < chunk_size;
           i_start += blockDim.x) {
        // load
        load_store(r_x, x, 0, i_start);
#pragma unroll
        for (int ii = 0; ii < kILP; ii++) {
          opmath_t next = static_cast<opmath_t>(r_x[ii]);
          if constexpr (norm_type == NormType::LInf) {
            vals[ii] = max_propagate_nan(vals[ii], ::abs(next));
          } else {
            vals[ii] += norm_type == NormType::L1 ? ::abs(next) : next * next;
          }
        }
      }
    } else {
      for (int64_t i_start = 0; i_start < n && i_start < chunk_size;
           i_start += blockDim.x * kILP) {
#pragma unroll
        for (int ii = 0; ii < kILP; ii++) {
          int i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            opmath_t next = static_cast<opmath_t>(x[i]);
            if constexpr (norm_type == NormType::LInf) {
              vals[ii] = max_propagate_nan(vals[ii], ::abs(next));
            } else {
              vals[ii] += norm_type == NormType::L1 ? ::abs(next) : next * next;
            }
          }
        }
      }
    }

    auto val = opmath_t(0);
    for (int i = 0; i < kILP; i++) {
      if constexpr (norm_type == NormType::LInf) {
        val = max_propagate_nan(val, vals[i]);
      } else {
        val += vals[i];
      }
    }
    auto final_val = norm_type == NormType::L1 || norm_type == NormType::L2
        ? at::native::cuda_utils::BlockReduceSum(val, s_vals)
        : at::native::cuda_utils::BlockReduceMax(val, s_vals);

    if (threadIdx.x == 0) {
      output_per_tensor
          [(tl.start_tensor_this_launch + tensor_loc) * max_chunks_per_tensor +
           chunk_idx] = final_val;
    }
  }
};

template <
    typename T,
    NormType norm_type,
    typename opmath_t = at::opmath_type<T>>
__global__ void lpnorm_cleanup(
    const opmath_t* output_per_tensor,
    TensorListAddresses addr_struct,
    int max_chunks_per_tensor) {
  __shared__ opmath_t vals[512];

  const opmath_t* output_this_tensor =
      output_per_tensor + blockIdx.x * max_chunks_per_tensor;
  opmath_t val = 0;
  for (size_t i = threadIdx.x; i < max_chunks_per_tensor; i += blockDim.x) {
    if constexpr (norm_type == NormType::LInf) {
      val = max_propagate_nan(val, output_this_tensor[i]);
    } else {
      val += output_this_tensor[i];
    }
  }
  opmath_t final_val = norm_type == NormType::L1 || norm_type == NormType::L2
      ? at::native::cuda_utils::BlockReduceSum<opmath_t>(val, vals)
      : at::native::cuda_utils::BlockReduceMax(val, vals);
  if (threadIdx.x == 0) {
    *(T*)addr_struct.addresses[blockIdx.x] =
        norm_type == NormType::L1 || norm_type == NormType::LInf
        ? final_val
        : ::sqrt(final_val);
  }
}

// note(mkozuki): Why excluding Int and Complex from fast path
// - Int: at::norm does not support.
// - Complex: __shfl_down_sync does not support complex and foreach does not
// support functions whose inputs dtypes and output dtype are different.
std::vector<Tensor> foreach_tensor_norm_cuda(
    TensorList tensors,
    const Scalar& ord) {
  double p;
  if (ord.isIntegral(false)) {
    p = ord.to<int64_t>();
  } else if (ord.isFloatingPoint()) {
    p = ord.to<double>();
  } else {
    TORCH_CHECK(
        false, "foreach_tensor_norm_cuda expects ord to be integer or float");
  }
  check_foreach_api_restrictions(tensors);
  const bool has_int_or_complex =
      std::any_of(tensors.begin(), tensors.end(), [](const auto& t) {
        const auto scalar_type = t.scalar_type();
        return at::isIntegralType(scalar_type, /*includeBool*/ true) ||
            at::isComplexType(scalar_type);
      });
  if (!can_use_fast_route(tensors) || has_int_or_complex ||
      !(p == static_cast<double>(1) || p == static_cast<double>(2) ||
        p == std::numeric_limits<double>::infinity())) {
    return foreach_tensor_norm_slow(tensors, ord);
  }

  const size_t ntensors = tensors.size();
  int max_chunks_per_tensor = -1;

  for (const auto t : c10::irange(ntensors)) {
    int max_chunks_this_tensor =
        (tensors[t].numel() + kChunkSize - 1) / kChunkSize;
    if (max_chunks_this_tensor > max_chunks_per_tensor) {
      max_chunks_per_tensor = max_chunks_this_tensor;
    }
  }
  const auto options = tensors[0].options();
  auto output_per_tensor = at::zeros(
      {static_cast<int64_t>(ntensors) * max_chunks_per_tensor},
      options.dtype(toOpMathType(tensors[0].scalar_type())));

  std::vector<at::Tensor> vec_res;
  vec_res.reserve(ntensors);
  for (const auto i : c10::irange(ntensors)) {
    vec_res.push_back(at::empty({}, options));
  }

  auto tensor_lists = std::vector<std::vector<Tensor>>{tensors.vec()};
  if (p == static_cast<double>(1)) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        kHalf,
        kBFloat16,
        tensor_lists[0][0].scalar_type(),
        "foreach_tensor_norm_cuda",
        [&]() {
          using opmath_t = typename at::opmath_type<scalar_t>;
          multi_tensor_apply<1>(
              tensor_lists,
              LpNormFunctor<scalar_t, NormType::L1>(),
              output_per_tensor.mutable_data_ptr<opmath_t>(),
              max_chunks_per_tensor);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          const at::cuda::OptionalCUDAGuard device_guard(
              device_of(output_per_tensor));
          auto stream = at::cuda::getCurrentCUDAStream();

          const size_t num_kernels = ceil_div(ntensors, MAX_TENSORS_PER_KERNEL);
          for (const auto i : c10::irange(num_kernels)) {
            const size_t num_tensors_this_kernel =
                (i < num_kernels - 1 || ntensors % MAX_TENSORS_PER_KERNEL == 0)
                ? MAX_TENSORS_PER_KERNEL
                : (ntensors % MAX_TENSORS_PER_KERNEL);

            TensorListAddresses addr_struct;
            for (const auto j : c10::irange(num_tensors_this_kernel)) {
              addr_struct.addresses[j] = vec_res[i * MAX_TENSORS_PER_KERNEL + j]
                                             .mutable_data_ptr<scalar_t>();
            }

            lpnorm_cleanup<scalar_t, NormType::L1>
                <<<num_tensors_this_kernel, 512, 0, stream>>>(
                    output_per_tensor.const_data_ptr<opmath_t>() +
                        i * MAX_TENSORS_PER_KERNEL * max_chunks_per_tensor,
                    addr_struct,
                    max_chunks_per_tensor);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }
        });
  } else if (p == static_cast<double>(2)) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        kHalf,
        kBFloat16,
        tensor_lists[0][0].scalar_type(),
        "foreach_tensor_norm_cuda",
        [&]() {
          using opmath_t = typename at::opmath_type<scalar_t>;
          multi_tensor_apply<1>(
              tensor_lists,
              LpNormFunctor<scalar_t, NormType::L2>(),
              output_per_tensor.mutable_data_ptr<opmath_t>(),
              max_chunks_per_tensor);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          const at::cuda::OptionalCUDAGuard device_guard(
              device_of(output_per_tensor));
          auto stream = at::cuda::getCurrentCUDAStream();

          const size_t num_kernels = ceil_div(ntensors, MAX_TENSORS_PER_KERNEL);
          for (const auto i : c10::irange(num_kernels)) {
            const size_t num_tensors_this_kernel =
                (i < num_kernels - 1 || ntensors % MAX_TENSORS_PER_KERNEL == 0)
                ? MAX_TENSORS_PER_KERNEL
                : (ntensors % MAX_TENSORS_PER_KERNEL);

            TensorListAddresses addr_struct;
            for (const auto j : c10::irange(num_tensors_this_kernel)) {
              addr_struct.addresses[j] = vec_res[i * MAX_TENSORS_PER_KERNEL + j]
                                             .mutable_data_ptr<scalar_t>();
            }

            lpnorm_cleanup<scalar_t, NormType::L2>
                <<<num_tensors_this_kernel, 512, 0, stream>>>(
                    output_per_tensor.const_data_ptr<opmath_t>() +
                        i * MAX_TENSORS_PER_KERNEL * max_chunks_per_tensor,
                    addr_struct,
                    max_chunks_per_tensor);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }
        });
  } else if (p == std::numeric_limits<double>::infinity()) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        kHalf,
        kBFloat16,
        tensor_lists[0][0].scalar_type(),
        "foreach_tensor_norm_cuda",
        [&]() {
          using opmath_t = typename at::opmath_type<scalar_t>;
          multi_tensor_apply<1>(
              tensor_lists,
              LpNormFunctor<scalar_t, NormType::LInf>(),
              output_per_tensor.mutable_data_ptr<opmath_t>(),
              max_chunks_per_tensor);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          const at::cuda::OptionalCUDAGuard device_guard(
              device_of(output_per_tensor));
          auto stream = at::cuda::getCurrentCUDAStream();

          const size_t num_kernels = ceil_div(ntensors, MAX_TENSORS_PER_KERNEL);
          for (const auto i : c10::irange(num_kernels)) {
            const size_t num_tensors_this_kernel =
                (i < num_kernels - 1 || ntensors % MAX_TENSORS_PER_KERNEL == 0)
                ? MAX_TENSORS_PER_KERNEL
                : (ntensors % MAX_TENSORS_PER_KERNEL);

            TensorListAddresses addr_struct;
            for (const auto j : c10::irange(num_tensors_this_kernel)) {
              addr_struct.addresses[j] = vec_res[i * MAX_TENSORS_PER_KERNEL + j]
                                             .mutable_data_ptr<scalar_t>();
            }

            lpnorm_cleanup<scalar_t, NormType::LInf>
                <<<num_tensors_this_kernel, 512, 0, stream>>>(
                    output_per_tensor.const_data_ptr<opmath_t>() +
                        i * MAX_TENSORS_PER_KERNEL * max_chunks_per_tensor,
                    addr_struct,
                    max_chunks_per_tensor);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }
        });
  } else {
    TORCH_CHECK(
        false,
        "foreach_tensor_norm_cuda fast path got unexpected ord value: ",
        p);
  }

  // correctly assign values to only non-empty slots, as the empty slots should
  // get skipped
  std::vector<Tensor> result;
  result.reserve(ntensors);
  int i = 0;
  for (const auto& t : tensors) {
    if (t.numel() != 0) {
      result.emplace_back(vec_res[i]);
      i++;
    } else {
      result.emplace_back(at::zeros({}, options));
    }
  }
  return result;
}

} // namespace at::native
