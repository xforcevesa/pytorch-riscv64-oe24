#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/Resize.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty.h>
#include <ATen/ops/split_with_sizes_copy_native.h>
#endif

namespace at::native {

namespace detail {

// NOTE [CUDA fast path for split_with_sizes_copy.out]
// split_with_sizes_copy.out for contiguous operands has the following
// properties:
// - Each src split consists of multiple chunks that are separated by a fixed
// stride. The number of chunks and the strides are the same across all src
// splits.
// - Each dst split is the concatenation of the chunks in its corresponding src
// splits.
// - The sizes of chunks vary across splits.
// - A (src, dst) chunk pair is not guaranteed to have the
// same alignment.
//
// The following strategies are employed to optimize for this workload:
// - The entire workload is fused into a single kernel to maximize I/O
// throughput and minimize wave quantization.
// - To account for both small and large chunk sizes, a "jagged grid" is used.
// Each chunk is processed by one or more blocks depending on its size.
// - Within each chunk, the region in which writes can be vectorized is
// identified. Within this region, writes are always vectorized and reads are
// oppurtunistically vectorized.
static constexpr int64_t BLOCK_SIZE = 128;
static constexpr int64_t BYTES_PER_THREAD = 16;

static __host__ __device__ inline int64_t div_up(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

template <typename T>
__device__ inline void stream_load128(uint4& val, const T* addr) {
  uint64_t low, high;
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  low = reinterpret_cast<const uint64_t*>(addr)[0];
  high = reinterpret_cast<const uint64_t*>(addr)[1];
#else
  asm("ld.global.nc.v2.u64 {%0, %1}, [%2];"
      : "=l"(low), "=l"(high)
      : "l"(addr));
#endif
  reinterpret_cast<uint64_t*>(&val)[0] = low;
  reinterpret_cast<uint64_t*>(&val)[1] = high;
}

template <typename T>
__device__ inline void stream_store128(T* addr, const uint4& val) {
  uint64_t low, high;
  low = reinterpret_cast<const uint64_t*>(&val)[0];
  high = reinterpret_cast<const uint64_t*>(&val)[1];
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  reinterpret_cast<uint64_t*>(addr)[0] = low;
  reinterpret_cast<uint64_t*>(addr)[1] = high;
#else
  asm("st.global.cs.v2.u64 [%0], {%1, %2};" : : "l"(addr), "l"(low), "l"(high));
#endif
}

template <typename T>
static __device__ inline bool is_aligned(const void* addr) {
  return reinterpret_cast<uintptr_t>(addr) % sizeof(T) == 0;
}

template <typename T>
static __device__ inline void load128(uint4& val, const char* addr) {
  for (size_t i = 0; i < detail::BYTES_PER_THREAD / sizeof(T); ++i) {
    reinterpret_cast<T*>(&val)[i] = reinterpret_cast<const T*>(addr)[i];
  }
}

template <>
__device__ inline void load128<uint4>(uint4& val, const char* addr) {
  stream_load128(val, addr);
}

static __device__ inline void load128(uint4& val, const char* addr) {
  if (is_aligned<uint4>(addr)) {
    load128<uint4>(val, addr);
  } else if (is_aligned<int64_t>(addr)) {
    load128<uint64_t>(val, addr);
  } else if (is_aligned<uint32_t>(addr)) {
    load128<uint32_t>(val, addr);
  } else {
    load128<uint8_t>(val, addr);
  }
}

static __device__ __inline__ void get_aligned_region(
    char* ptr,
    const int64_t chunk_size,
    const int64_t alignment,
    int64_t& align_off,
    int64_t& aligned_size) {
  const int64_t ptr_val = reinterpret_cast<uintptr_t>(ptr);
  align_off = detail::div_up(ptr_val, alignment) * alignment - ptr_val;
  aligned_size = (chunk_size - align_off) / alignment * alignment;
}

static __device__ __inline__ void copy_chunk(
    char* dst,
    const char* src,
    int64_t chunk_size,
    int64_t thread_idx,
    int64_t num_threads) {
  if (chunk_size < num_threads) {
    if (thread_idx < chunk_size) {
      dst[thread_idx] = src[thread_idx];
    }
    return;
  }

  // Identify the region in which writes are guaranteed to be 128-bit aligned
  int64_t align_off, aligned_size;
  get_aligned_region(
      dst, chunk_size, detail::BYTES_PER_THREAD, align_off, aligned_size);

  for (int64_t off = align_off + thread_idx * detail::BYTES_PER_THREAD;
       off < align_off + aligned_size;
       off += num_threads * detail::BYTES_PER_THREAD) {
    uint4 val;
    // Oppurtunistically vectorize reads
    load128(val, &src[off]);
    stream_store128(&dst[off], val);
  }

  // Handle unaligned regions
  if (thread_idx < align_off && thread_idx < chunk_size) {
    dst[thread_idx] = src[thread_idx];
  }
  if (align_off + aligned_size + thread_idx < chunk_size) {
    dst[align_off + aligned_size + thread_idx] =
        src[align_off + aligned_size + thread_idx];
  }
}

static __global__ void split_with_sizes_copy_out_contiguous_no_cast_kernel(
    char** dst_base_addrs,
    char** src_base_addrs,
    int64_t* split_chunk_sizes,
    int64_t* block_idx_to_split_idx,
    int64_t* blocks_cumsums,
    int64_t src_stride,
    int64_t num_chunks) {
  const int64_t split_idx = block_idx_to_split_idx[blockIdx.x];
  const int64_t split_blocks =
      blocks_cumsums[split_idx + 1] - blocks_cumsums[split_idx];
  const int64_t split_threads = split_blocks * blockDim.x;
  const int64_t split_thread_idx =
      (blockIdx.x - blocks_cumsums[split_idx]) * blockDim.x + threadIdx.x;
  const int64_t split_chunk_size = split_chunk_sizes[split_idx];

  char* dst_base_addr = dst_base_addrs[split_idx];
  char* src_base_addr = src_base_addrs[split_idx];

  for (int64_t i = blockIdx.y; i < num_chunks; i += gridDim.y) {
    copy_chunk(
        dst_base_addr + i * split_chunk_size,
        src_base_addr + i * src_stride,
        split_chunk_size,
        split_thread_idx,
        split_threads);
  }
}

// Calculate the base addr for each split.
static inline std::vector<int64_t> get_split_base_addrs(
    const at::Tensor& tensor,
    at::IntArrayRef split_sizes,
    int64_t dim) {
  const auto* data_ptr = static_cast<char*>(tensor.data_ptr());
  const auto strides = tensor.strides();
  const auto element_sz = tensor.element_size();
  int64_t off = 0;
  std::vector<int64_t> split_base_addrs;
  split_base_addrs.reserve(split_sizes.size());
  for (const auto& split_size : split_sizes) {
    split_base_addrs.push_back(reinterpret_cast<int64_t>(data_ptr + off));
    off += split_size * strides[dim] * element_sz;
  }
  return split_base_addrs;
}

static inline std::vector<int64_t> get_dst_addrs(at::TensorList out) {
  std::vector<int64_t> addrs;
  addrs.reserve(out.size());
  for (const auto& tensor : out) {
    addrs.push_back(reinterpret_cast<int64_t>(tensor.data_ptr()));
  }
  return addrs;
}

// Calculate the chunk size for each split in bytes.
static inline std::vector<int64_t> get_split_chunk_sizes(
    const at::Tensor& tensor,
    at::IntArrayRef split_sizes,
    int64_t dim) {
  const auto stride = tensor.stride(dim);
  const auto element_sz = tensor.element_size();
  std::vector<int64_t> split_chunk_sizes;
  split_chunk_sizes.reserve(split_sizes.size());
  for (const auto& split_size : split_sizes) {
    split_chunk_sizes.push_back(split_size * stride * element_sz);
  }
  return split_chunk_sizes;
}

// Calculate the chunk stride in bytes. This is the same for all splits.
static inline int64_t get_chunk_stride(const at::Tensor& tensor, int64_t dim) {
  int64_t stride = 1;
  for (int64_t d = dim; d < tensor.dim(); ++d) {
    stride *= tensor.sizes()[d];
  }
  return stride * tensor.element_size();
}

// Calculate the number of chunks. This is the same for all splits.
static inline int64_t get_num_chunks(const at::Tensor& tensor, int64_t dim) {
  int64_t num_chunks = tensor.numel();
  for (int64_t d = dim; d < tensor.dim(); ++d) {
    num_chunks /= tensor.sizes()[d];
  }
  return num_chunks;
}

// Pack multiple std::vector<int64_t> into a single cuda tensor.
std::pair<at::Tensor, std::vector<int64_t*>> pack_vecs(
    std::vector<const std::vector<int64_t>*> vecs,
    const at::Device& device) {
  int64_t numel = 0;
  for (const auto* vec : vecs) {
    numel += vec->size();
  }

  auto packed = at::empty(
      {numel}, at::TensorOptions().dtype(at::kLong).pinned_memory(true));
  size_t offset = 0;
  for (const auto* vec : vecs) {
    memcpy(
        packed.data_ptr<int64_t>() + offset,
        vec->data(),
        sizeof(int64_t) * vec->size());
    offset += vec->size();
  }
  packed = packed.to(device, /*non_blocking=*/true);

  std::vector<int64_t*> ptrs;
  ptrs.reserve(vecs.size());
  offset = 0;
  for (const auto* vec : vecs) {
    ptrs.push_back(packed.data_ptr<int64_t>() + offset);
    offset += vec->size();
  }
  return std::make_pair(std::move(packed), std::move(ptrs));
}

} // namespace detail

// See [CUDA fast path for split_with_sizes_copy.out]
void split_with_sizes_copy_out_cuda_contiguous_no_cast(
    const at::Tensor& self,
    at::IntArrayRef split_sizes,
    int64_t dim,
    at::TensorList out) {
  const auto device = self.device();
  const auto src_base_addrs =
      detail::get_split_base_addrs(self, split_sizes, dim);
  const auto dst_base_addrs = detail::get_dst_addrs(out);
  const auto src_stride = detail::get_chunk_stride(self, dim);
  const auto split_chunk_sizes =
      detail::get_split_chunk_sizes(self, split_sizes, dim);
  const auto num_chunks = detail::get_num_chunks(self, dim);

  // Calculate the number of blocks required for the first chunk across all
  // splits, assuming each thread only processes BYTES_PER_THREAD bytes.
  int64_t num_blocks = 0;
  for (const auto& split_chunk_size : split_chunk_sizes) {
    num_blocks += detail::div_up(
        split_chunk_size, detail::BLOCK_SIZE * detail::BYTES_PER_THREAD);
  }

  // Calculate the maximum number of blocks to launch. Only consider
  // maxThreadsPerMultiProcessor as a limiting factor as the kernel uses no
  // shared memory and little registers. Over-subscribe the SMs to hide I/O
  // latency.
  const auto num_sms =
      at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  const auto max_threads_per_sm =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor;
  const int64_t max_blocks =
      num_sms * max_threads_per_sm / detail::BLOCK_SIZE * 2.0;

  // Make each thread process BYTES_PER_THREAD * iter_factor bytes to regulate
  // block size. Spread iter_factor evenly between chunks_per_block and
  // iters_per_chunk.
  int64_t iter_factor = detail::div_up(num_blocks * num_chunks, max_blocks);
  int64_t chunks_per_block = std::ceil(std::sqrt(iter_factor));
  chunks_per_block = std::min(chunks_per_block, num_chunks);
  const int64_t iters_per_chunk = detail::div_up(iter_factor, chunks_per_block);

  // Launch a logically jagged grid of shape
  // (chunk_size*, num_splits, num_chunks / chunks_per_block)
  // backed by a physical grid of shape
  // (sum(chunk_size), num_chunks / chunks_per_block).
  // A block can find its split_idx via block_idx_to_split_idx.
  std::vector<int64_t> block_idx_to_split_idx;
  std::vector<int64_t> blocks_cumsums{0};
  block_idx_to_split_idx.reserve(num_blocks);
  for (size_t split_idx = 0; split_idx < split_sizes.size(); ++split_idx) {
    const auto blocks = detail::div_up(
        split_chunk_sizes[split_idx],
        detail::BLOCK_SIZE * detail::BYTES_PER_THREAD * iters_per_chunk);
    block_idx_to_split_idx.insert(
        block_idx_to_split_idx.end(), blocks, split_idx);
    blocks_cumsums.push_back(blocks_cumsums.back() + blocks);
  }

  dim3 blocks(blocks_cumsums.back(), num_chunks / chunks_per_block, 1);
  dim3 threads(detail::BLOCK_SIZE, 1, 1);

  auto [_, ptrs] = detail::pack_vecs(
      {&dst_base_addrs,
       &src_base_addrs,
       &split_chunk_sizes,
       &block_idx_to_split_idx,
       &blocks_cumsums},
      device);

  detail::split_with_sizes_copy_out_contiguous_no_cast_kernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      /*dst_base_addrs=*/reinterpret_cast<char**>(ptrs[0]),
      /*src_base_addrs=*/reinterpret_cast<char**>(ptrs[1]),
      /*split_chunk_sizes=*/ptrs[2],
      /*block_idx_to_split_idx=*/ptrs[3],
      /*blocks_cumsums=*/ptrs[4],
      src_stride,
      num_chunks);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void split_with_sizes_copy_out_cuda(
    const Tensor& self,
    IntArrayRef split_sizes,
    int64_t dim,
    TensorList out) {
  bool contiguous_no_cast = self.is_non_overlapping_and_dense();
  for (const auto& t : out) {
    contiguous_no_cast &= t.is_non_overlapping_and_dense();
    contiguous_no_cast &= (t.dtype() == self.dtype());
  }
  if (contiguous_no_cast) {
    // Perform equivalent checks performed by the composite impl
    if (dim < 0) {
      dim = at::maybe_wrap_dim(dim, self.dim());
    }
    TORCH_CHECK(
        self.dim() != 0, "split expects at least a 1-dimensional tensor")

    const int64_t dim_size = self.size(dim);
    int64_t split_sizes_sum = 0;
    for (const auto i : c10::irange(split_sizes.size())) {
      TORCH_CHECK(
          split_sizes[i] >= 0,
          "split_with_sizes expects split_sizes have only non-negative ",
          "entries, but got split_sizes=",
          split_sizes[i]);
      split_sizes_sum += split_sizes[i];
    }
    TORCH_CHECK(
        split_sizes_sum == dim_size,
        "split_with_sizes expects split_sizes to sum exactly to ",
        dim_size,
        " (input tensor's size at dimension ",
        dim,
        "), ",
        "but got split_sizes=",
        split_sizes);

    TORCH_CHECK(
        out.size() == split_sizes.size(),
        "split_with_sizes_copy_out() expected an out= argument of size ",
        split_sizes.size(),
        ", got size ",
        out.size());

    auto out_shape = self.sizes().vec();
    for (const auto i : c10::irange(split_sizes.size())) {
      out_shape[dim] = split_sizes[i];
      if (resize_output_check(out[i], out_shape)) {
        out[i].resize_(out_shape);
      }
      TORCH_CHECK(
          out[i].dtype() == self.dtype(),
          "Expected out tensor to have dtype ",
          self.dtype(),
          ", but got ",
          out[i].dtype(),
          " instead");
      TORCH_CHECK(
          out[i].device() == self.device(),
          "Expected out tensor to have device ",
          self.device(),
          ", but got ",
          out[i].device(),
          " instead");
    }
    split_with_sizes_copy_out_cuda_contiguous_no_cast(
        self, split_sizes, dim, out);
  } else {
    at::native::split_with_sizes_copy_out(self, split_sizes, dim, out);
  }
}

} // namespace at::native
